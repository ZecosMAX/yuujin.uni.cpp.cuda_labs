﻿#define SAVE_BMP_IMPLEMENT
#include "common.cuh"
#include "labs_tasks.cuh"


int main(int argc, char* argv[])
{
    bool isZludaRuntime = CheckArgument(argc, argv, "zluda");
    if (!Initialize(isZludaRuntime, argc, argv))
        return -1;
    
    RunAllTasks();

    return 0;
}

void RunAllTasks()
{
    //Lab2_Task1();
    //Lab2_Task2();
    //Lab2_Task3();

    Lab3_Task1();
    Lab3_Task2();

    Lab4_Task1();
}
